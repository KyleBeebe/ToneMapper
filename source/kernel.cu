#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <string>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/copy.h>

#include "ToneMapper.h"
#define STB_IMAGE_IMPLEMENTATION
#define STBI_NO_FAILURE_STRINGS
#include "stb_image.h"

struct RGB{
	stbi_uc r;
	stbi_uc g;
	stbi_uc b;
};

__global__ void ToIntesity(stbi_uc* d_pixel_data, stbi_uc* d_intensities)
{
	//extern __shared__ stbi_uc s_pixel_data[];
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	stbi_uc r = d_pixel_data[idx];
}

__host__ void Run(const Image& img)
{
	int total_pixels = img.mHeight * img.mWidth;
	int num_blocks_x = std::ceil(img.mWidth / 16.0f);
	int num_blocks_y = std::ceil(img.mHeight / 16.0f);

	dim3 grid(num_blocks_x, num_blocks_y);
	dim3 block(16, 16);

    ToIntesity<<<grid, block>>>(nullptr, nullptr);
}

int main()
{
	int nDevices{0}, activeDevice{-1};
	hipDeviceProp_t devProps;
	ToneMapper toneMapper; 
	std::string imgFilepath{"images/roses.jpg"};

	if (hipGetDeviceCount(&nDevices) != hipSuccess || !nDevices)
	{
		std::cerr << "No Cuda Device Detected!" << std::endl;
		return 1;
	}

	for (int i = 0; i < nDevices; ++i) {
		hipGetDeviceProperties(&devProps,i);
		if (devProps.major >= 6) 
		{
			activeDevice = i;
			break;
		}
	}

	if (activeDevice < 0)
	{
		std::cerr << "No Cuda Device with Compute Capability 6.X+ Detected!" << std::endl;
		return 1;
	}

	try 
	{
		toneMapper.Load(imgFilepath);
	}
	catch (const ImageDoesntExistException& e)
	{
		std::cerr << e.what() << "\n";
	}
	
	Run(toneMapper.GetImage());

    return 0;
}

